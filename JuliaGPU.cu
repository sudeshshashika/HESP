#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <complex>
#include <chrono>
#include <hip/hip_runtime.h>
#include "lodepng.h"
#include "Complex.h"

namespace lodepng {
unsigned encode(const std::string& filename,
const unsigned char* image,
unsigned w, unsigned h,
LodePNGColorType colortype,
unsigned bitdepth);  
}


__global__ void Julia(double* X, double* Y, double* image, unsigned char* ArImg, Complex c_,int N){
    
    long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    long long idy = blockIdx.y * blockDim.y + threadIdx.y;
    if((idx<N)&&(idy<N)){
            int iters = 0;                     //iteration count                 
            Complex z(X[idy],Y[idx]);          //definign the complex number
            while(iters<250){
                z = z * z + c_;
                if(z.absolute_val(z) <100)
                {
                    image[idx*N+idy] = 10*iters%255; 
                }
                ++iters;
                
                ArImg[(idx*N+idy)*3]=image[idx*N+idy];
            }
    }
    __syncthreads();
}

int main(){

    //domain size and mapping in HOST
    double domain      = 4.0;                   //betweev -2 and +2
    double pixel_width = 2048;                 //2048
    double dx          = (domain/pixel_width)*1;
    int N              = (domain/dx); 
    int Arraay_size    = N*N*3;               //this is for png encoding
    Complex c_ (-0.8,0.2);                      //defining const complex number for Julia
    int nBytes         = N*sizeof(double);
    int n2DBytes       = N*N*sizeof(double);

    double* X=nullptr; double* Y=nullptr; double* image=nullptr; unsigned char* ArImg=nullptr;
    X     = (double*)malloc(nBytes);
    Y     = (double*)malloc(nBytes);
    image = (double*)malloc(n2DBytes);
    ArImg = (unsigned char*)malloc(Arraay_size);

    //initialization vectors
    for(int i=0;i<N;++i){
        X[i] = -2+i*dx;
        Y[i] = -2+i*dx;                 //2-i*dx
    }

    double* d_X;
    double* d_Y;
    double* d_image;
    unsigned char* d_ArImg;

    hipMalloc(&d_X, nBytes);
    hipMalloc(&d_Y, nBytes);
    hipMalloc(&d_image, n2DBytes);
    hipMalloc(&d_ArImg, Arraay_size);


    hipMemcpy(d_X, X, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, Y, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_image, image, n2DBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_ArImg, ArImg, Arraay_size, hipMemcpyHostToDevice);
    
    int threads_per_block = 32.0;
    int number_of_blocks  = (int)ceil((double)N/threads_per_block);
    auto START = std::chrono::system_clock::now();
    Julia<<< number_of_blocks, threads_per_block >>>(d_X, d_Y, d_image, d_ArImg, c_, N);
    hipDeviceSynchronize();
    auto END   = std::chrono::system_clock::now();
    std::chrono::duration<double>elepsed_time = END-START;

    hipMemcpy(ArImg, d_ArImg, Arraay_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // For Encoding png
    const char* filename = "min0802i.png";
    unsigned width       = 2048, height = 2048;
    using lodepng::encode;
    encode(filename, ArImg, width, height,LCT_RGB,8);
    printf("RUN_TIME = %f sec. \n",elepsed_time.count());


    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_image);
    hipFree(d_ArImg);

    free (X);
    free (Y);
    free (image);
    free (ArImg);
    return 0;
}